
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 32

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    __shared__ float M[TILE_SIZE][TILE_SIZE];
    __shared__ float N[TILE_SIZE][TILE_SIZE];
	
    int tx,ty,row,col;
    float Cv=0.0;
    
    tx=threadIdx.x;
    ty=threadIdx.y;
    row=blockIdx.y*blockDim.y+threadIdx.y;
    col=blockIdx.x*blockDim.x+threadIdx.x;
	
    for(int i=0; i<(k-1)/TILE_SIZE+1; ++i)
    {
      if(i*TILE_SIZE+tx<k && row<m)
	    M[ty][tx]=A[row*k+i*TILE_SIZE+tx];
      else
	    M[ty][tx]=0.0; 
	    
      if(i*TILE_SIZE+ty<k&&col<n)
	    N[ty][tx]=B[col+(i*TILE_SIZE+ty)*n];
      else
	    N[ty][tx]=0.0;
	    
     __syncthreads();
	    
    for(int j=0; j<TILE_SIZE; ++j)
	    Cv+=M[ty][j]*N[j][tx];
	    
    __syncthreads();
  
    }
	
    if(row<m&&col<n)
	    C[row*n+col]=Cv;
	
    /*************************************************************************/
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE
    dim3 dimGrid((n-1)/BLOCK_SIZE+1,(m-1)/BLOCK_SIZE+1,1);
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE,1);
    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
    mysgemm <<< dimGrid, dimBlock >>>(m, n, k, A, B, C);
    /*************************************************************************/
}


