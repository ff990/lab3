/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    __shared__ float ds_M[TILE_SIZE][TILE_SIZE];
    __shared__ float ds_N[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    int Row = by * blockDim.y + ty;
    int Col = bx * blockDim.x + tx;
    float Cvalue = 0.0;

    // Loop over the M and N tiles required to compute the P element
    for (int p = 0; p < (k-1)/TILE_SIZE+1; ++p) {
        // Collaborative loading of M and N tiles into shared memory

        if (Row<m && p*TILE_SIZE+tx<k)
            ds_M[ty][tx] = A[Row*k + p*TILE_SIZE+tx];
        else
            ds_M[ty][tx] = 0.0;
        if (p*TILE_SIZE+ty<k && Col<n)
            ds_N[ty][tx] = B[(p*TILE_SIZE+ty)*n + Col];
        else
            ds_N[ty][tx] = 0.0;
        __syncthreads();

        for (int i=0; i<TILE_SIZE; ++i) Cvalue += ds_M[ty][i] * ds_N[i][tx];
        __syncthreads();
    }
    if (Row<m && Col<n) C[Row*n+Col] = Cvalue;    
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C)
{
    

    // Initialize thread block and kernel grid dimensions ---------------------


    //INSERT CODE HERE
    dim3 dimGrid((n-1)/TILE_SIZE + 1, (m-1)/TILE_SIZE + 1, 1);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);



    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE
    mysgemm <<< dimGrid, dimBlock >>>(m, n, k, A, B, C);



}
